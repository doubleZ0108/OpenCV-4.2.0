#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "array.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "kernel_dispatcher.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

#include <cstddef>
#include <vector>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, std::size_t Rank>
        __global__ void permute(
            array<index_type, Rank> axis_order,
            Span<T> output, array<size_type, Rank> outStrides,
            View<T> input, array<size_type, Rank> inStrides)
        {
            for (auto i : grid_stride_range(input.size())) {
                index_type oldPosition = 0;
                index_type newPosition = i;

                for (int j = 0; j < Rank; j++)
                {
                    auto order = axis_order[j];
                    oldPosition += (newPosition / outStrides[j]) * inStrides[order];
                    newPosition %= outStrides[j];
                }

                output[i] = input[oldPosition];
            }
        }
    }

    template <class T, std::size_t Rank> static
    void launch_permute_kernel(
        const Stream& stream,
        const std::vector<std::size_t>& order,
        Span<T> output, const std::vector<std::size_t>& outStride,
        View<T> input, const std::vector<std::size_t>& inStride)
    {
        CV_Assert(order.size() == Rank);
        CV_Assert(outStride.size() == Rank);
        CV_Assert(inStride.size() == Rank);

        array<index_type, Rank> order_k;
        order_k.assign(std::begin(order), std::end(order));

        array<size_type, Rank> outStride_k, inStride_k;
        outStride_k.assign(std::begin(outStride), std::end(outStride));
        inStride_k.assign(std::begin(inStride), std::end(inStride));

        auto kernel = raw::permute<T, Rank>;
        auto policy = make_policy(kernel, input.size(), 0, stream);
        launch_kernel(kernel, policy, order_k, output, outStride_k, input, inStride_k);
    }

    GENERATE_KERNEL_DISPATCHER(permute_dispatcher, launch_permute_kernel);

    template <class T>
    void permute(
        const Stream& stream,
        TensorSpan<T> output, TensorView<T> input,
        std::vector<std::size_t> order)
    {
        CV_Assert(output.rank() == input.rank());
        CV_Assert(input.rank() == order.size());
        CV_Assert(input.size() == output.size());

        /* squeezable axes at the beginning of both tensors which aren't permuted can be eliminated
         *
         * Reasoning:
         * ----------
         * Suppose an item's indices in the input tensor is [i1, i2, ...]. The indices in the
         * output tensor will be some permutation of the input tensor indices. Let the output
         * tensor indices be [o1, o2, ...]. The permutation operation essentially copies items
         * from the input tensor to new locations in the output tensor as dictated by the indices.
         *
         * If the size of the first axis of the input and output tensor is one and these axes are
         * not involved in any permutation, i.e. order[0] = 0, the input and output indicies for
         * all the elements will be of the form be [0, i2, ...] and [0, o2, ...] respectively.
         * The first index does not contribute to the element's address calculation and hence does
         * nothing apart from eating up few cycles.
         */
        while (order[0] == 0 && input.get_axis_size(0) == 1 && output.get_axis_size(0) == 1) {
            /* remove the axes */
            input.squeeze(0);
            output.squeeze(0);

            /* when we remove axis zero, the axis index will be one less than the previous index
             * for the remaining axes
             */
            order.erase(order.begin());
            for (auto& axis : order)
                axis--;

            /* optimizations should not break the invariants */
            CV_Assert(output.rank() == input.rank());
            CV_Assert(input.rank() == order.size());
            CV_Assert(input.size() == output.size());
        }

        auto rank = output.rank();
        auto inShape = input.shape_as_vector();
        auto outShape = output.shape_as_vector();

        std::vector<std::size_t> inStride(rank), outStride(rank);
        inStride.back() = 1;
        outStride.back() = 1;
        /* garbage, ..., garbage, 1 */

        std::copy(std::begin(inShape) + 1, std::end(inShape), std::begin(inStride));
        std::copy(std::begin(outShape) + 1, std::end(outShape), std::begin(outStride));
        /* dim[0], dim[1], ..., dim[-1], 1 */

        std::partial_sum(inStride.rbegin(), inStride.rend(), inStride.rbegin(), std::multiplies<std::size_t>());
        std::partial_sum(outStride.rbegin(), outStride.rend(), outStride.rbegin(), std::multiplies<std::size_t>());
        /* stride[0], stride[1], ..., stride[-2], 1 */

        CV_Assert(2 <= rank && rank <= CSL_MAX_TENSOR_RANK);
        permute_dispatcher<T, 2, CSL_MAX_TENSOR_RANK>(rank, stream, order, output, outStride, input, inStride);
    }

    template void permute(const Stream&, TensorSpan<__half>, TensorView<__half>, std::vector<std::size_t>);
    template void permute(const Stream&, TensorSpan<float>, TensorView<float>, std::vector<std::size_t>);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
