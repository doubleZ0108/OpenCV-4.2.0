#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "limits.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {

        template <class T>
        __global__ void roi_pooling(
            Span<T> output, size_type pooled_height, size_type pooled_width,
            View<T> input, size_type in_height, size_type in_width,
            View<T> rois, size_type num_channels, T spatial_scale)
        {
            // input: [1, num_channels, in_height, in_width]
            // rois: [num_rois, 5]

            // output: [num_rois, num_channels, pooled_height, pooled_width]
            const auto out_spatial_size = pooled_height * pooled_width;
            const auto out_roi_size = num_channels * out_spatial_size;

            /* every element in the output is mapped to a window in the input and each thread processes several windows */
            for (auto idx : grid_stride_range(output.size()))
            {
                const auto n = idx / out_roi_size;
                const auto c = (idx % out_roi_size) / out_spatial_size;
                const auto y = (idx % out_spatial_size) / pooled_width;
                const auto x = idx % pooled_width;

                const index_type roi_offset = n * 5;

                using device::round;
                const index_type batch_id = rois[roi_offset + 0];
                const index_type x_start_roi = round(rois[roi_offset + 1] * spatial_scale);
                const index_type y_start_roi = round(rois[roi_offset + 2] * spatial_scale);
                const index_type x_end_roi = round(rois[roi_offset + 3] * spatial_scale);
                const index_type y_end_roi = round(rois[roi_offset + 4] * spatial_scale);

                using device::max;
                const auto roi_width = max<index_type>(x_end_roi - x_start_roi + 1, 1);
                const auto roi_height = max<index_type>(y_end_roi - y_start_roi + 1, 1);

                const auto roi_width_ratio = static_cast<T>(roi_width) / static_cast<T>(pooled_width);
                const auto roi_height_ratio = static_cast<T>(roi_height) / static_cast<T>(pooled_height);

                auto x_start = x_start_roi + static_cast<index_type>(static_cast<T>(x) * roi_width_ratio);
                auto y_start = y_start_roi + static_cast<index_type>(static_cast<T>(y) * roi_height_ratio);

                using device::ceil;
                auto x_end = x_start_roi + static_cast<index_type>(ceil(static_cast<T>(x + 1) * roi_width_ratio));
                auto y_end = y_start_roi + static_cast<index_type>(ceil(static_cast<T>(y + 1) * roi_height_ratio));

                using device::max;
                x_start = max<index_type>(x_start, 0);
                y_start = max<index_type>(y_start, 0);

                using device::min;
                x_end = min<index_type>(x_end, in_width);
                y_end = min<index_type>(y_end, in_height);

                /* We have to set the output to zero if (x_start >= x_end) or (y_start >= y_end). If either
                 * condition is true, the loops below won't execute even a single iteration. Hence, by setting
                 * `max_val` to zero in this case, we can combine it with the `else` code.
                 */
                T max_val = (x_start >= x_end || y_start >= y_end) ? T(0) : device::numeric_limits<T>::lowest();

                const index_type in_offset = (batch_id * num_channels + c) * in_height * in_width;
                for (auto iy = y_start; iy < y_end; iy++)
                {
                    for (auto ix = x_start; ix < x_end; ix++)
                    {
                        const auto in_idx = in_offset + iy * in_width + ix;
                        max_val = max(max_val, input[in_idx]);
                    }
                }

                output[idx] = max_val;
            }
        }
    }

    template <class T>
    void roi_pooling(const Stream& stream, TensorSpan<T> output, TensorView<T> input, View<T> rois, T spatial_scale)
    {
        CV_Assert(input.get_axis_size(1) == output.get_axis_size(1));

        size_type num_channels = output.get_axis_size(1);

        size_type pooled_height = output.get_axis_size(2);
        size_type pooled_width = output.get_axis_size(3);

        size_type in_height = input.get_axis_size(2);
        size_type in_width = input.get_axis_size(3);

        auto kernel = raw::roi_pooling<T>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, pooled_height, pooled_width, input, in_height, in_width, rois, num_channels, spatial_scale);
    }

    template void roi_pooling(const Stream& stream, TensorSpan<__half> output, TensorView<__half> input, View<__half> rois, __half spatial_scale);
    template void roi_pooling(const Stream& stream, TensorSpan<float> output, TensorView<float> input, View<float> rois, float spatial_scale);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
