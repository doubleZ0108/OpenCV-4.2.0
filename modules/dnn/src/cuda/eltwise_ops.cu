#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"
#include "vector_traits.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <opencv2/core.hpp>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T, std::size_t N>
        __global__ void eltwise_max_2_vec(Span<T> output, View<T> x, View<T> y) {
            using vector_type = get_vector_type_t<T, N>;

            auto output_vPtr = vector_type::get_pointer(output.data());
            auto x_vPtr = vector_type::get_pointer(x.data());
            auto y_vPtr = vector_type::get_pointer(y.data());

            for (auto i : grid_stride_range(output.size() / vector_type::size())) {
                vector_type vec_x, vec_y;
                v_load(vec_x, x_vPtr[i]);
                v_load(vec_y, y_vPtr[i]);

                for (int j = 0; j < vector_type::size(); j++) {
                    using device::max;
                    vec_x.data[j] = max(vec_x.data[j], vec_y.data[j]);
                }

                v_store(output_vPtr[i], vec_x);
            }
        }

        template <class T, std::size_t N>
        __global__ void eltwise_sum_2_vec(Span<T> output, View<T> x, View<T> y) {
            using vector_type = get_vector_type_t<T, N>;

            auto output_vPtr = vector_type::get_pointer(output.data());
            auto x_vPtr = vector_type::get_pointer(x.data());
            auto y_vPtr = vector_type::get_pointer(y.data());

            for (auto i : grid_stride_range(output.size() / vector_type::size())) {
                vector_type vec_x, vec_y;
                v_load(vec_x, x_vPtr[i]);
                v_load(vec_y, y_vPtr[i]);

                for (int j = 0; j < vector_type::size(); j++)
                    vec_x.data[j] = vec_x.data[j] + vec_y.data[j];

                v_store(output_vPtr[i], vec_x);
            }
        }

        template <class T, std::size_t N>
        __global__ void eltwise_sum_coeff_2_vec(Span<T> output, T coeff_x, View<T> x, T coeff_y, View<T> y) {
            using vector_type = get_vector_type_t<T, N>;

            auto output_vPtr = vector_type::get_pointer(output.data());
            auto x_vPtr = vector_type::get_pointer(x.data());
            auto y_vPtr = vector_type::get_pointer(y.data());

            for (auto i : grid_stride_range(output.size() / vector_type::size())) {
                vector_type vec_x, vec_y;
                v_load(vec_x, x_vPtr[i]);
                v_load(vec_y, y_vPtr[i]);

                for (int j = 0; j < vector_type::size(); j++)
                    vec_x.data[j] = coeff_x * vec_x.data[j] + coeff_y * vec_y.data[j];

                v_store(output_vPtr[i], vec_x);
            }
        }

        template <class T, std::size_t N>
        __global__ void eltwise_prod_2_vec(Span<T> output, View<T> x, View<T> y) {
            using vector_type = get_vector_type_t<T, N>;

            auto output_vPtr = vector_type::get_pointer(output.data());
            auto x_vPtr = vector_type::get_pointer(x.data());
            auto y_vPtr = vector_type::get_pointer(y.data());

            for (auto i : grid_stride_range(output.size() / vector_type::size())) {
                vector_type vec_x, vec_y;
                v_load(vec_x, x_vPtr[i]);
                v_load(vec_y, y_vPtr[i]);

                for (int j = 0; j < vector_type::size(); j++)
                    vec_x.data[j] = vec_x.data[j] * vec_y.data[j];

                v_store(output_vPtr[i], vec_x);
            }
        }

        template <class T, std::size_t N>
        __global__ void eltwise_div_2_vec(Span<T> output, View<T> x, View<T> y) {
            using vector_type = get_vector_type_t<T, N>;

            auto output_vPtr = vector_type::get_pointer(output.data());
            auto x_vPtr = vector_type::get_pointer(x.data());
            auto y_vPtr = vector_type::get_pointer(y.data());

            for (auto i : grid_stride_range(output.size() / vector_type::size())) {
                vector_type vec_x, vec_y;
                v_load(vec_x, x_vPtr[i]);
                v_load(vec_y, y_vPtr[i]);

                for (int j = 0; j < vector_type::size(); j++)
                    vec_x.data[j] = vec_x.data[j] / vec_y.data[j];

                v_store(output_vPtr[i], vec_x);
            }
        }
    }

    template <class T, std::size_t N>
    void launch_vectorized_eltwise_max_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(is_fully_aligned<T>(output, N));
        CV_Assert(is_fully_aligned<T>(x, N));
        CV_Assert(is_fully_aligned<T>(y, N));

        auto kernel = raw::eltwise_max_2_vec<T, N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, x, y);
    }

    template <class T>
    void eltwise_max_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(x.size() == y.size());
        CV_Assert(x.size() == output.size());

        if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
            launch_vectorized_eltwise_max_2<T, 4>(stream, output, x, y);
        } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 2)) {
            launch_vectorized_eltwise_max_2<T, 2>(stream, output, x, y);
        } else {
            launch_vectorized_eltwise_max_2<T, 1>(stream, output, x, y);
        }
    }

    template void eltwise_max_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_max_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);

    template <class T, std::size_t N>
    void launch_vectorized_eltwise_sum_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(is_fully_aligned<T>(output, N));
        CV_Assert(is_fully_aligned<T>(x, N));
        CV_Assert(is_fully_aligned<T>(y, N));

        auto kernel = raw::eltwise_sum_2_vec<T, N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, x, y);
    }

    template <class T>
    void eltwise_sum_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(x.size() == y.size());
        CV_Assert(x.size() == output.size());

        if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
            launch_vectorized_eltwise_sum_2<T, 4>(stream, output, x, y);
        } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 2)) {
            launch_vectorized_eltwise_sum_2<T, 2>(stream, output, x, y);
        } else {
            launch_vectorized_eltwise_sum_2<T, 1>(stream, output, x, y);
        }
    }

    template void eltwise_sum_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_sum_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);

    template <class T, std::size_t N>
    void launch_vectorized_eltwise_sum_coeff_2(const Stream& stream, Span<T> output, T coeff_x, View<T> x, T coeff_y, View<T> y) {
        CV_Assert(is_fully_aligned<T>(output, N));
        CV_Assert(is_fully_aligned<T>(x, N));
        CV_Assert(is_fully_aligned<T>(y, N));

        auto kernel = raw::eltwise_sum_coeff_2_vec<T, N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, coeff_x, x, coeff_y, y);
    }

    template <class T>
    void eltwise_sum_coeff_2(const Stream& stream, Span<T> output, T coeff_x, View<T> x, T coeff_y, View<T> y) {
        CV_Assert(x.size() == y.size());
        CV_Assert(x.size() == output.size());

        if (static_cast<float>(coeff_x) == 1.0f && static_cast<float>(coeff_y) == 1.0f) {
            eltwise_sum_2(stream, output, x, y);
            return;
        }

        if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
            launch_vectorized_eltwise_sum_coeff_2<T, 4>(stream, output, coeff_x, x, coeff_y, y);
        } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 2)) {
            launch_vectorized_eltwise_sum_coeff_2<T, 2>(stream, output, coeff_x, x, coeff_y, y);
        } else {
            launch_vectorized_eltwise_sum_coeff_2<T, 1>(stream, output, coeff_x, x, coeff_y, y);
        }
    }

    template void eltwise_sum_coeff_2(const Stream&, Span<__half>, __half, View<__half>, __half, View<__half>);
    template void eltwise_sum_coeff_2(const Stream&, Span<float>, float, View<float>, float, View<float>);

    template <class T, std::size_t N>
    void launch_vectorized_eltwise_prod_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(is_fully_aligned<T>(output, N));
        CV_Assert(is_fully_aligned<T>(x, N));
        CV_Assert(is_fully_aligned<T>(y, N));

        auto kernel = raw::eltwise_prod_2_vec<T, N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, x, y);
    }

    template <class T>
    void eltwise_prod_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(x.size() == y.size());
        CV_Assert(x.size() == output.size());

        if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
            launch_vectorized_eltwise_prod_2<T, 4>(stream, output, x, y);
        } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 2)) {
            launch_vectorized_eltwise_prod_2<T, 2>(stream, output, x, y);
        } else {
            launch_vectorized_eltwise_prod_2<T, 1>(stream, output, x, y);
        }
    }

    template void eltwise_prod_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_prod_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);

    template <class T, std::size_t N>
    void launch_vectorized_eltwise_div_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(is_fully_aligned<T>(output, N));
        CV_Assert(is_fully_aligned<T>(x, N));
        CV_Assert(is_fully_aligned<T>(y, N));

        auto kernel = raw::eltwise_div_2_vec<T, N>;
        auto policy = make_policy(kernel, output.size() / N, 0, stream);
        launch_kernel(kernel, policy, output, x, y);
    }

    template <class T>
    void eltwise_div_2(const Stream& stream, Span<T> output, View<T> x, View<T> y) {
        CV_Assert(x.size() == y.size());
        CV_Assert(x.size() == output.size());

        if (is_fully_aligned<T>(output, 4) && is_fully_aligned<T>(x, 4) && is_fully_aligned<T>(y, 4)) {
            launch_vectorized_eltwise_div_2<T, 4>(stream, output, x, y);
        } else if (is_fully_aligned<T>(output, 2) && is_fully_aligned<T>(x, 2) && is_fully_aligned<T>(y, 2)) {
            launch_vectorized_eltwise_div_2<T, 2>(stream, output, x, y);
        } else {
            launch_vectorized_eltwise_div_2<T, 1>(stream, output, x, y);
        }
    }

    template void eltwise_div_2(const Stream& stream, Span<__half> output, View<__half> x, View<__half> y);
    template void eltwise_div_2(const Stream& stream, Span<float> output, View<float> x, View<float> y);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
